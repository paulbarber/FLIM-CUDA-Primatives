/* FLIM-CUDA-Primatives
 * P Barber <paul.barber@oncology.ox.ac.uk>, 2017
 * Sum the time data at each pixel to produce an intensity image
 * GPU code.
 */

// includes CUDA
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
/*! Kernel to sum the time data at each pixel to produce an intensity image
 *
 * @param g_idata   input data in global memory
 * @param g_odata   output data in global memory
 * @param ntimepts  number of time points in a transient - the first dimension of the data
 * @param width     width of image - the second dimension of the data
 * @param height    height of image - the third dimension of the data
 */
__global__ void
CudaKernel(float *g_idata, float *g_odata, size_t ntimepts, size_t width, size_t height)
{
	// we should create one thread per pixel, and one thread block per row of the image
	int y = blockIdx.x;    // The block indicates the row
    int x = threadIdx.x;   // The thread in the block indicates the col
	
	// TODO - Code here :-)
	// This may have one thread per entry in the output array (256), and each one strides through the data in global mem adding them together
	// OR it may have one thread per pixel, and each one takes its lifetime data and adds it to the output array in shared memory, need to use atomic adds.
}

////////////////////////////////////////////////////////////////////////////////
/*! Wrapper function to run the kernel from cpu c code.
 */
extern "C" void 
executeCudaKernel(unsigned int threadsPerBlock, unsigned int  blocksPerGrid, float *d_idata, float *d_odata, size_t ntimepts, size_t width, size_t height)
{
	CudaKernel<<<blocksPerGrid, threadsPerBlock>>>(d_idata, d_odata, ntimepts, width, height);
}