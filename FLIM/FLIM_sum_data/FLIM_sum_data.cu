/* FLIM-CUDA-Primatives
 * P Barber <paul.barber@oncology.ox.ac.uk>, 2017
 * Sum the time data at each pixel to produce an intensity image
 * GPU code.
 */

// includes CUDA
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
/*! Kernel to sum the time data at each pixel to produce an intensity image
 *
 * @param g_idata   input data in global memory
 * @param g_odata   output data in global memory
 * @param ntimepts  number of time points in a transient - the first dimention of the data
 * @param width     width of image - the second dimention of the data
 * @param height    height of image - the third dimention of the data
 */
__global__ void
CudaKernel(float *g_idata, float *g_odata, size_t ntimepts, size_t width, size_t height)
{
	// we should create one thread per pixel, and one thread block per row of the image
	int y = blockIdx.x;    // The block indicates the row
    int x = threadIdx.x;   // The thread in the block indicates the col
	
	//size_t pixel_pitch = g_idata->pitch;
	//size_t width_pitch = pixel_pitch * width;
	//char *row_start = (char*)g_idata->ptr + y * width_pitch;  // is char* because width_pitch is in bytes
	//float *trans = (float*)(row_start + x * pixel_pitch);

	float *row_start = g_idata + y * width;
	float *trans = row_start + x;

	float sum = 0.0;
	for(size_t t=0; t<ntimepts; ++t){
		sum += *trans;
		trans++;
	}
	
	g_odata[y*width + x] = sum;
}

////////////////////////////////////////////////////////////////////////////////
/*! Wrapper function to run the kernel from cpu c code.
 */
extern "C" void 
executeCudaKernel(unsigned int threadsPerBlock, unsigned int  blocksPerGrid, float *d_idata, float *d_odata, size_t ntimepts, size_t width, size_t height)
{
	CudaKernel<<<blocksPerGrid, threadsPerBlock>>>(d_idata, d_odata, ntimepts, width, height);
}