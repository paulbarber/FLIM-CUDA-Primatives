#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

extern "C"
void computeGold(float *reference, float *idata, size_t ntimepts, size_t width, size_t height);

// Macro to time CPU operations
clock_t time_start, time_msec, time_diff;
#define time_this(a) time_start=clock(); a; time_diff = clock() - time_start; time_msec = time_diff * 1000 / CLOCKS_PER_SEC; printf("%s: %d ms\n", #a, time_msec);


////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(float *g_idata, float *g_odata, size_t ntimepts, size_t width, size_t height)
{
	// we are creating one thread per pixel, and one thread block per row of the image
	int y = blockIdx.x;    // The block indicates the row
    int x = threadIdx.x;   // The thread in the block indicates the col
	
	//size_t pixel_pitch = g_idata->pitch;
	//size_t width_pitch = pixel_pitch * width;
	//char *row_start = (char*)g_idata->ptr + y * width_pitch;  // is char* because width_pitch is in bytes
	//float *trans = (float*)(row_start + x * pixel_pitch);

	float *row_start = g_idata + y * width;
	float *trans = row_start + x;

	float sum = 0.0;
	for(size_t t=0; t<ntimepts; ++t){
		sum += *trans;
		trans++;
	}
	
	g_odata[y*width + x] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

	// setup some CUDA timers
	StopWatchInterface *hostToDeviceTimer = 0;
	StopWatchInterface *timer = 0;
	StopWatchInterface *DeviceToHostTimer = 0;
    sdkCreateTimer(&hostToDeviceTimer);
    sdkCreateTimer(&timer);
    sdkCreateTimer(&DeviceToHostTimer);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    size_t width = 128;
    size_t height = 128;
    size_t ntimepts = 256;
	size_t mem_size = height * width * ntimepts * sizeof(float);
	size_t image_size = height * width * sizeof(float);
	
    // allocate host memory
	// data from ics file will be t,x,y (contigious data is t - height (y) has largest stride)
	// keep 3d data format the same (in cuda 3D contigious data is w - then h - d=depth has the largest stride)
//	hipPitchedPtr h_idata = make_hipPitchedPtr(malloc(mem_size), ntimepts*sizeof(float), ntimepts, width); 
	float *h_idata = (float*)malloc(mem_size); 

    // initalize the memory, using the recommend way to access hipPitchedPtr data
//	size_t pixel_pitch = h_idata.pitch;
//	size_t width_pitch = pixel_pitch * width;
	for (size_t y = 0; y < height; ++y)
    {
//		char *row_start = (char*)h_idata.ptr + y * width_pitch;  // is char* because width_pitch is in bytes
		float *row_start = h_idata + y * width * ntimepts;
		for (unsigned int x = 0; x < width; ++x)
		{
//			float *time = (float*)(row_start + x * pixel_pitch);
			float *time = row_start + x * ntimepts;
			for (unsigned int t = 0; t < ntimepts; ++t)
			{
				time[t] = (float)(255-t);
			}
		}
	}

    sdkStartTimer(&hostToDeviceTimer);

    // allocate device memory
//    hipPitchedPtr d_idata;
//    hipExtent extent = make_hipExtent(ntimepts*sizeof(float), width, height);
//    checkCudaErrors(hipMalloc3D(&d_idata, extent));
	float *d_idata;
	checkCudaErrors(hipMalloc(&d_idata, mem_size));

	// copy host memory to device
	//hipMemcpy3DParms memcpy3DParms = {0};
	//memcpy3DParms.srcPtr = h_idata;
	//memcpy3DParms.dstPtr = d_idata;
	//memcpy3DParms.kind = hipMemcpyHostToDevice;
	//memcpy3DParms.extent = extent;
	////memcpy3DParms.srcPos = make_hipPos(0,0,0);
	////memcpy3DParms.dstPos = make_hipPos(0,0,0);
 //   checkCudaErrors(hipMemcpy3D(&memcpy3DParms));
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

	// allocate device memory for result
    float *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_odata, image_size));

    sdkStopTimer(&hostToDeviceTimer);
    sdkStartTimer(&timer);

    // setup execution parameters
	size_t threadsPerBlock = width;
    size_t blocksPerGrid = (width*height) / threadsPerBlock;
    testKernel<<<blocksPerGrid, threadsPerBlock>>>(d_idata, d_odata, ntimepts, width, height);

    sdkStopTimer(&timer);
    sdkStartTimer(&DeviceToHostTimer);
	
    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(image_size);

    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_odata, d_odata, image_size, hipMemcpyDeviceToHost));

    sdkStopTimer(&DeviceToHostTimer);

    // Report timings
	printf("GPU:\n");
	printf("Host To Device time: %f (ms)\n", sdkGetTimerValue(&hostToDeviceTimer));
	printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
	printf("Device To Host time: %f (ms)\n", sdkGetTimerValue(&DeviceToHostTimer));
    sdkDeleteTimer(&hostToDeviceTimer);
    sdkDeleteTimer(&timer);
    sdkDeleteTimer(&DeviceToHostTimer);

    // compute reference solution
    float *reference = (float *) malloc(image_size);
	printf("CPU:\n");
	time_this(computeGold(reference, h_idata, ntimepts, width, height));

    // check result
    //if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
    //{
    //    // write file for regression test
    //    sdkWriteFile("./data/regression.dat", h_odata, num_threads, 0.0f, false);
    //}
    //else
    //{
    //    // custom output handling when no regression test running
    //    // in this case check if the result is equivalent to the expected soluion
    //    bTestResult = compareData(reference, h_odata, num_threads, 0.0f, 0.0f);
    //}

    // cleanup memory
	free(h_idata);
    free(h_odata);
    free(reference);
	checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits   
    hipDeviceReset();
 
	printf("Press a key to finish.\n");
	getc(stdin);
	
	exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
